#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cassert>
#include <time.h>

#include "matrix.h"
#include "utils.h"
#include "graph.h"

using namespace std;

void testMatrix()
{
	srand((unsigned)(time(NULL)));
	int row = 10000, column = 100;
	int max = 1000;
	CSR<float> csr;
	GenerateSparseMatrix(csr, row, column, 0.9, max);
	for (int i = 0; i < csr.data_num_; i++)
	{
		assert(csr.data[i] < max);
	}
	Matrix<float> matrix(row, column);
	matrix.InitWithRandom(max);
	for (int i = 0; i < matrix.row_; i++)
	{
		for (int j = 0; j < matrix.column_; j++)
		{
			assert(matrix.data_[i * column + j] < max);
		}
	}
}
void testGraph()
{
	CSRCOOGraph<float> graph("/home/xinger/cpp/cuda/cuda-programming/cuda_examples/common/bin/test.txt");
}
int main()
{
	testGraph();
	return 0;
}