#include <iostream>
#include <string>

#include "src/kernel.cuh"
#include "common/graph.h"
#include "common/utils.h"

using namespace std;
int main()
{
	string filename = "/home/xinger/cpp/cuda/cuda-programming/cuda_examples/common/bin/test.txt";
	CSRGraph<float> graph(filename);

	unsigned edge_num = graph.GetEdgeNum();

	// copy data to gpu
	graph.ToDevice();
	// graph.OutputGraph();
	unsigned *d_result;
	unsigned result = 0;
	hipMalloc(&d_result, sizeof(unsigned));
	hipMemcpy(d_result, &result, sizeof(unsigned), hipMemcpyHostToDevice);
	dim3 blockSize(256);
	dim3 gridSize((edge_num + blockSize.x - 1) / blockSize.x);
	double t1 = getCurrentTime();
	// launch kernel
	triangleCountLaunch(graph, d_result, gridSize, blockSize);
	hipDeviceSynchronize();
	double t2 = getCurrentTime();
	cout << "time: " << t2 - t1 << " ms" << endl;
	// unsigned result;
	hipMemcpy(&result, d_result, sizeof(unsigned), hipMemcpyDeviceToHost);

	cout << "triangle count: " << result / 6 << endl;
	hipFree(d_result);
}