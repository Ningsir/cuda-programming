#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <time.h>

#include "common/matrix.h"
#include "common/utils.h"
#include "common/cuda_helper.cuh"
#include "src/kernel.cuh"

using namespace std;

int main()
{
	srand((unsigned)(time(NULL)));
	unsigned int m = 8000, n = 80, k = 1000, max = 100;
	// matrix mul: w * matrix_a
	// matrix w
	Matrix<float> w(m, n);
	w.InitWithMostZero(0.9, max);
	CSR<float> csr;
	Matrix2CSR(csr, w);
	// matrix a
	Matrix<float> matrix_a(n, k);
	matrix_a.InitWithRandom(max);

	unsigned *d_row_ptr, *d_column_index;
	float *d_w, *d_a, *d_o;
	checkCudaErrors(hipMalloc(&d_row_ptr, sizeof(unsigned) * csr.row_num_));
	hipMalloc(&d_column_index, sizeof(unsigned) * csr.data_num_);
	hipMalloc(&d_w, sizeof(float) * csr.data_num_);
	hipMalloc(&d_a, sizeof(float) * matrix_a.row_ * matrix_a.column_);
	hipMalloc(&d_o, sizeof(float) * m * k);

	hipMemcpy(d_row_ptr, csr.row_ptr, sizeof(unsigned) * csr.row_num_, hipMemcpyHostToDevice);
	hipMemcpy(d_column_index, csr.col, sizeof(unsigned) * csr.data_num_, hipMemcpyHostToDevice);
	hipMemcpy(d_w, csr.data, sizeof(unsigned) * csr.data_num_, hipMemcpyHostToDevice);
	hipMemcpy(d_a, matrix_a.data_, sizeof(unsigned) * matrix_a.row_ * matrix_a.column_, hipMemcpyHostToDevice);

	dim3 blockSize(8, 8);
	dim3 gridSize((csr.row_num_ - 1 + blockSize.x - 1) / blockSize.x,
				  (matrix_a.column_ + blockSize.y - 1) / blockSize.y);
	sparseMatrixMulLaunch<float>(gridSize, blockSize, d_row_ptr, d_column_index, d_w, d_a, d_o, m, n, k);
	// 检查内核参数错误
	checkCudaErrors(hipPeekAtLastError());
	hipDeviceSynchronize();

	float *output = (float *)(malloc(sizeof(float) * m * k));
	hipMemcpy(output, d_o, sizeof(float) * m * k, hipMemcpyDeviceToHost);

	// dense matrix mul
	float *d_A, *d_B, *d_C;
	hipMalloc(&d_A, sizeof(float) * m * n);
	hipMalloc(&d_B, sizeof(float) * n * k);
	hipMalloc(&d_C, sizeof(float) * m * k);

	hipMemcpy(d_A, w.data_, sizeof(float) * m * n, hipMemcpyHostToDevice);
	hipMemcpy(d_B, matrix_a.data_, sizeof(float) * matrix_a.row_ * matrix_a.column_, hipMemcpyHostToDevice);
	matrixMul(gridSize, blockSize, d_A, d_B, d_C, m, n, k);
	hipDeviceSynchronize();
	float *output1 = (float *)(malloc(sizeof(float) * m * k));
	hipMemcpy(output1, d_C, sizeof(float) * m * k, hipMemcpyDeviceToHost);

	for (int i = 0; i < m * k; i++)
	{
		assert(output[i] == output1[i]);
	}
	free(output);
	free(output1);
	hipFree(d_row_ptr);
	hipFree(d_column_index);
	hipFree(d_w);
	hipFree(d_a);
	hipFree(d_o);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	return 0;
}