#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <time.h>

#include "common/matrix.h"
#include "common/utils.h"

using namespace std;

template <typename T>
__global__ void sparseMatrixMul(unsigned int *row_ptr, unsigned int *col_index,
								T *w, T *a, T *o, unsigned int m, unsigned int n, unsigned int k)
{
	int row = blockDim.x * blockIdx.x + threadIdx.x;
	int col = blockDim.y * blockIdx.y + threadIdx.y;

	int start = row_ptr[row];
	int end = row_ptr[row + 1];

	T sum = 0;
	for (int i = 0; i < end - start; i++)
	{
		sum += w[start + i] * a[col_index[start + i] * k + col];
	}
	o[row * k + col] = sum;
}

int main()
{
	srand((unsigned)(time(NULL)));
	int m = 80, n = 80, k = 80, max = 100;
	CSR<float> csr;
	GenerateSparseMatrix(csr, m, n, 0.9, max);
	Matrix<float> matrix;
	GenerateMatrix(matrix, n, k, max);

	unsigned *d_row_ptr, *d_column_index;
	float *d_w, *d_a, *d_o;
	hipMalloc(&d_row_ptr, sizeof(unsigned) * csr.row_num_);
	hipMalloc(&d_column_index, sizeof(unsigned) * csr.data_num_);
	hipMalloc(&d_w, sizeof(float) * csr.data_num_);
	hipMalloc(&d_a, sizeof(float) * matrix.row_ * matrix.column_);
	hipMalloc(&d_o, sizeof(float) * m * k);

	hipMemcpy(d_row_ptr, csr.row_ptr, sizeof(unsigned) * csr.row_num_, hipMemcpyHostToDevice);
	hipMemcpy(d_column_index, csr.col, sizeof(unsigned) * csr.data_num_, hipMemcpyHostToDevice);
	hipMemcpy(d_w, csr.data, sizeof(unsigned) * csr.data_num_, hipMemcpyHostToDevice);
	hipMemcpy(d_a, matrix.data_, sizeof(unsigned) * matrix.row_ * matrix.column_, hipMemcpyHostToDevice);

	dim3 blockSize(32, 32);
	dim3 gridSize((csr.row_num_ - 1 + blockSize.x - 1) / blockSize.x,
				  (matrix.column_ + blockSize.y - 1) / blockSize.y);
	sparseMatrixMul<<<gridSize, blockSize>>>(d_row_ptr, d_column_index, d_w, d_a, d_o, m, n, k);
	hipDeviceSynchronize();

	float *output = (float *)(malloc(sizeof(float) * m * k));
	hipMemcpy(output, d_o, sizeof(unsigned) * m * k, hipMemcpyDeviceToHost);
	for (int i = 0; i < m * k; i++)
	{
		cout << output[i] << ", ";
	}
	free(output);
	hipFree(d_row_ptr);
	hipFree(d_column_index);
	hipFree(d_w);
	hipFree(d_a);
	hipFree(d_o);
	return 0;
}