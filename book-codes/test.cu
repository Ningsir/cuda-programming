
#include <hip/hip_runtime.h>
#include <iostream>

struct Matrix
{
    int width;
    int height;
    float *elements;
};

__device__ float getElement(Matrix *A, int row, int col)
{
    return A->elements[row * A->width + col];
}

__device__ void setElement(Matrix *A, int row, int col, float value)
{
    A->elements[row * A->width + col] = value;
}

__global__ void matMulKernel(Matrix *A, Matrix *B, Matrix *C)
{
    float Cvalue = 0.0;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = 0; i < A->width; ++i)
    {
        Cvalue += getElement(A, row, i) * getElement(B, i, col);
    }
    setElement(C, row, col, Cvalue);
}

int main()
{
    int width = 32;
    int height = 32;
    Matrix *A, *B, *C;

    hipMallocManaged((void **)&A, sizeof(Matrix));
    hipMallocManaged((void **)&B, sizeof(Matrix));
    hipMallocManaged((void **)&C, sizeof(Matrix));
    int nBytes = width * height * sizeof(float);
    hipMallocManaged((void **)&A->elements, nBytes);
    hipMallocManaged((void **)&B->elements, nBytes);
    hipMallocManaged((void **)&C->elements, nBytes);

    A->height = height;
    A->width = width;
    B->height = height;
    B->width = width;
    C->height = height;
    C->width = width;
    for (int i = 0; i < width * height; ++i)
    {
        A->elements[i] = 1.0;
        B->elements[i] = 2.0;
    }

    dim3 blockSize(32, 32);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                  (height + blockSize.y - 1) / blockSize.y);

    matMulKernel<<<gridSize, blockSize>>>(A, B, C);

    hipDeviceSynchronize();
    for (int i = 0; i < width; i++)
    {
        for (int j = 0; j < height; j++)
        {
            std::cout << C->elements[i * width + j] << " ";
        }
        std::cout << std::endl;
    }
    float maxError = 0.0;
    for (int i = 0; i < width * height; ++i)
        maxError = fmax(maxError, fabs(C->elements[i] - 2 * width));
    std::cout << "maxError: " << maxError << std::endl;

    return 0;
}
